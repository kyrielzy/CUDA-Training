
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}
void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;
  for(int i = index; i < N; i+=stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  //a = (float *)malloc(size);
  checkCuda(hipMallocManaged(&a,size));
  //b = (float *)malloc(size);
  checkCuda(hipMallocManaged(&b,size));
  //c = (float *)malloc(size);
  checkCuda(hipMallocManaged(&c,size));
  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);
  size_t threadOfPerBlock = 256;
  size_t numOfBlock = (N + threadOfPerBlock - 1) / threadOfPerBlock;
  addVectorsInto<<<numOfBlock,threadOfPerBlock>>>(c, a, b, N);

  //addVectorsInto(c, a, b, N);
  checkCuda(hipGetLastError());
  checkCuda(hipDeviceSynchronize());

  checkElementsAre(7, c, N);

  checkCuda(hipFree(a));
  checkCuda(hipFree(b));
  checkCuda(hipFree(c));
}
